
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Function Definitions */
double getRandom(double min, double max);
__global__ void transpose(double *h_a, double *d_a, int dimA);

int main(int arc, char* argv[]){

    int i;       // Loop Counter
    double *h_a;    // Host Memory Pointer
    double *d_a;    // Device Memory Pointer
    int dimA = 4096 * 4096; // Size of Array
    size_t memSize; // Memory Size
    double gigaflops;   // Gigaflops Store
    /* Thread Hierarchy Variables */
    int nBlocks = 1;
    int tPB = 1024;
    /* Time Variables */
    time_t timer1, timer2;
    timer1 = time(NULL);

    /* Allocate Memory */
    memSize = dimA * sizeof(double);
    h_a = (double*) malloc(memSize);
    hipMalloc((void**) &d_a, memSize);

    /* Populate Matrix */
    srand(time(NULL));
    for(i = 0; i < dimA; i++){
        h_a[i] = getRandom(1.0, 2.0);
    }

    /* Copy to Device */
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    /* Launch Kernel */
    dim3 dimGrid(nBlocks);
    dim3 dimBlock(tPB);
    transpose<<<dimGrid,dimBlock>>>(d_a, h_a, dimA);

    /* Copy Result */
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);

    /* Stop Clocks */
    timer2 = time(NULL);

    gigaflops = ((dimA * 4960.0 * 6.0) / 1000000000.0) / difftime(timer2, timer1);
    printf("Estimated GFlops/sec: %lf\n", gigaflops);
}
    
double getRandom(double min, double max){
    return (max - min) * ((double)rand() / (double)RAND_MAX) + min;
}

__global__ void transpose(double *d_a, double *h_a, int dimA){
    int i, j, k;
    i = blockIdx.x;
    j = threadIdx.x;
    int spot = (i * dimA) + j;
    for (k = 0; k < dimA; k++){
        d_a[spot] += h_a[(i * dimA) + k] * h_a[(k * dimA) + j];
    }
}